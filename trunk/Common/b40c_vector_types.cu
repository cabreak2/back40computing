/**
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 */

/******************************************************************************
 * Utility code for working with vector types of arbitary typenames
 ******************************************************************************/

#pragma once
#include <hip/hip_runtime.h>


namespace b40c {


/**
 * Specializations of this vector-type template can be used to indicate the 
 * proper vector type for a given typename and vector size. We can use the ::Type
 * typedef to declare and work with the appropriate vectorized type for a given 
 * typename T.
 * 
 * For example, consider the following copy kernel that uses vec-2 loads 
 * and stores:
 * 
 * 		template <typename T>
 * 		__global__ void CopyKernel(T *d_in, T *d_out) 
 * 		{
 * 			typename VecType<T, 2>::Type datum;
 * 
 * 			typename VecType<T, 2>::Type *d_in_v2 = (typename VecType<T, 2>::Type *) d_in;
 * 			typename VecType<T, 2>::Type *d_out_v2 = (typename VecType<T, 2>::Type *) d_out;
 * 
 * 			datum = in[threadIdx.x];
 * 			out[threadIdx.x] = datum;
 * 		} 
 * 
 * We can use this kernel to copy int streams using 64-bit loads, e.g.:  
 * 
 * 		CopyKernel<int><<<1, 128>>>(d_in, d_out) 
 *    
 */
template <typename T, int vec_elements> struct VecType;

/**
 * Partially-specialized generic vec1 type 
 */
template <typename T> 
struct VecType<T, 1> {
	T x;
	typedef T Type;
};

/**
 * Partially-specialized generic vec2 type 
 */
template <typename T> 
struct VecType<T, 2> {
	T x;
	T y;
	typedef VecType<T, 2> Type;
};

/**
 * Partially-specialized generic vec4 type 
 */
template <typename T> 
struct VecType<T, 4> {
	T x;
	T y;
	T z;
	T w;
	typedef VecType<T, 4> Type;
};


/**
 * Macro for expanding partially-specialized built-in vector types
 */
#define B40C_DEFINE_VECTOR_TYPE(base_type,short_type)                           \
  template<> struct VecType<base_type, 1> { typedef short_type##1 Type; };      \
  template<> struct VecType<base_type, 2> { typedef short_type##2 Type; };      \
  template<> struct VecType<base_type, 4> { typedef short_type##4 Type; };     

B40C_DEFINE_VECTOR_TYPE(char,               char)
B40C_DEFINE_VECTOR_TYPE(signed char,        char)
B40C_DEFINE_VECTOR_TYPE(short,              short)
B40C_DEFINE_VECTOR_TYPE(int,                int)
B40C_DEFINE_VECTOR_TYPE(long,               long)
B40C_DEFINE_VECTOR_TYPE(long long,          longlong)
B40C_DEFINE_VECTOR_TYPE(unsigned char,      uchar)
B40C_DEFINE_VECTOR_TYPE(unsigned short,     ushort)
B40C_DEFINE_VECTOR_TYPE(unsigned int,       uint)
B40C_DEFINE_VECTOR_TYPE(unsigned long,      ulong)
B40C_DEFINE_VECTOR_TYPE(unsigned long long, ulonglong)
B40C_DEFINE_VECTOR_TYPE(float,              float)
B40C_DEFINE_VECTOR_TYPE(double,             double)

#undef B40C_DEFINE_VECTOR_TYPE


} // namespace b40c

